#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"

#define blockSize 128

namespace StreamCompaction {
    namespace Efficient {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }

        __global__ void kernUpSweep(int n, int d, int* idata) {
            int idx = blockIdx.x * blockDim.x + threadIdx.x;
            int k = (1 << (d + 1)) * idx;
            if (k >= n) return;

            idata[k + 1 << (d + 1) - 1] += idata[k + 1 << d - 1];
        }

        __global__ void kernDownSweep(int n, int d, int* idata) {
            int idx = blockIdx.x * blockDim.x + threadIdx.x;
            int k = (1 << (d + 1)) * idx;
            if (k >= n) return;

            int t = idata[k + 1 << d - 1];
            idata[k + 1 << d - 1] = idata[k + 1 << (d + 1) - 1];
            idata[k + 1 << (d + 1) - 1] += t;
        }

        __global__ void kernToExclusive(int n, int* odata, int* idata) {
            int idx = blockIdx.x * blockDim.x + threadIdx.x;
            if (idx >= n) return;

            if (idx == 0) {
                odata[idx] = 0;
            }
            else {
                odata[idx] = idata[idx - 1];
            }
            return;
        }

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata, bool time) {
            int log2CeilN = ilog2ceil(n);
            int n_padded = 1 << log2CeilN;

            int* dev_odata;
            int* dev_idata;

            hipMalloc((void**)&dev_odata, n_padded * sizeof(int));
            checkCUDAError("hipMalloc dev_odata failed");
            hipMalloc((void**)&dev_idata, n_padded * sizeof(int));
            checkCUDAError("hipMalloc dev_idata failed");

            // pad to zero for entries >= n
            hipMemset(dev_odata, 0, n_padded * sizeof(int));
            checkCUDAError("hipMemset dev_odata failed");
            hipMemset(dev_idata, 0, n_padded * sizeof(int));
            checkCUDAError("hipMemset dev_idata failed");

            hipMemcpy(dev_idata, idata, n * sizeof(int), hipMemcpyHostToDevice);
            checkCUDAError("hipMemcpy to dev_idata failed");

            if (time)
                timer().startGpuTimer();
            // TODO

            // upsweep
            for (int d = 0; d < log2CeilN; d++) {
                // "for all k = 0 to n-1 by 2^(d+1) in parallel"
                dim3 blocksPerGrid((n_padded / (1 << (d + 1)) + blockSize - 1) / blockSize);
                kernUpSweep<<<blocksPerGrid, blockSize>>>(n, d, dev_idata);
            }

            // set root to zero
            hipMemset(dev_idata + n_padded - 1, 0, sizeof(int));

            // downseep
            for (int d = log2CeilN - 1; d >= 0; d--) {
                // "for all k = 0 to n-1 by 2^(d+1) in parallel"
                dim3 blocksPerGrid((n_padded / (1 << (d + 1)) + blockSize - 1) / blockSize);
                kernDownSweep<<<blocksPerGrid, blockSize>>>(n, d, dev_idata);
            }

            //dim3 blocksPerGrid((n + blockSize - 1) / blockSize);
            //kernToExclusive<<<blocksPerGrid, blockSize>>>(n, dev_odata, dev_idata);

            if (time)
                timer().endGpuTimer();

            hipMemcpy(odata, dev_odata, n * sizeof(int), hipMemcpyDeviceToHost);
            checkCUDAError("hipMemcpy from dev_odata failed");

            hipFree(dev_odata);
            checkCUDAError("hipFree dev_odata failed");
            hipFree(dev_idata);
            checkCUDAError("hipFree dev_idata failed");
        }

        /**
         * Performs stream compaction on idata, storing the result into odata.
         * All zeroes are discarded.
         *
         * @param n      The number of elements in idata.
         * @param odata  The array into which to store elements.
         * @param idata  The array of elements to compact.
         * @returns      The number of elements remaining after compaction.
         */
        int compact(int n, int *odata, const int *idata) {
            timer().startGpuTimer();
            // TODO
            timer().endGpuTimer();
            return -1;
        }
    }
}
